// TODO: Efficient Checking in Interpolation
// Function Names: functionName
// Variable Name: variable_name
// Filtering values not matching


#include <hip/hip_runtime.h>
#include<bits/stdc++.h>
#include <chrono>

using namespace std;

int pad = 255;

// x --> x-coordinate, y --> y-coordinate
__device__
bool inLimits(int x, int y, int x_max, int y_max){
	if(x >= 0 && x < x_max && y >= 0 && y < y_max){
		return true;
	}
	return false;
}

__device__
float sine(int angle) {
	if(angle == 0){
		return 0;
	}
	else if(angle == 45){
		return 0.70710678118;
	}
	return -0.70710678118;
}

__device__
float cosine(int angle) {
	if(angle == 0){
		return 1;
	}
	return 0.70710678118;
}

// point[0] --> x-coordinate, point[1] --> y-coordinate
// pivot[0] --> x-coordinate, pivot[1] --> y-coordinate
// res[0] --> x-coordinate, res[1] --> y-coordinate
__device__
void getRotatedCoordinate(float *res, int *point, int *pivot, int angle) {
	float s = sine(angle);
	float c = cosine(angle);

	float x_new = (point[0] - pivot[0])*c - (point[1] - pivot[1])*s;
	float y_new = (point[0] - pivot[0])*s + (point[1] - pivot[1])*c;

	res[0] = x_new + pivot[0];
	res[1] = y_new + pivot[1];
}

float sineHost(int angle) {
	if(angle == 0){
		return 0;
	}
	else if(angle == 45){
		return 1/sqrt(2);
	}
	return -1*(1/sqrt(2));
}

float cosineHost(int angle) {
	if(angle == 0){
		return 1;
	}
	return 1/sqrt(2);
}

// point[0] --> x-coordinate, point[1] --> y-coordinate
// pivot[0] --> x-coordinate, pivot[1] --> y-coordinate
// res[0] --> x-coordinate, res[1] --> y-coordinate

void getRotatedCoordinateHost(float *res, int point[], int pivot[], int angle) {
	float s = sineHost(angle);
	float c = cosineHost(angle);

	float x_new = (point[0] - pivot[0])*c - (point[1] - pivot[1])*s;
	float y_new = (point[0] - pivot[0])*s + (point[1] - pivot[1])*c;

	res[0] = x_new + pivot[0];
	res[1] = y_new + pivot[1];
}

__device__
void interpolate(int *X, int point[], int pivot[], int angle, float *interpolated, int x_m, int x_n){
	float v[2];
	getRotatedCoordinate(v, point, pivot, angle);
	float x = v[0], y = v[1];
	int x1 = floor(v[0]), y1 = floor(v[1]); // corresponds to (0, 0)
	int x2 = floor(v[0]), y2 = floor(v[1] + 1); // corresponds to (0, 1)
	int x3 = floor(v[0] + 1), y3 = floor(v[1]); // corresponds to (1, 0)
	int x4 = floor(v[0] + 1), y4 = floor(v[1] + 1); // corresponds to (1, 1)
	if(!inLimits(x1, y1, x_n, x_m) || !inLimits(x4, y4, x_n, x_m)){
		interpolated[0] = -1;
		return;
	}
	for(int i = 0; i < 3; i++){
		float z00 = X[y1*x_n*3 + x1*3 + i];
		float z01 = X[y2*x_n*3 + x2*3 + i];
		float z10 = X[y3*x_n*3 + x3*3 + i];
		float z11 = X[y4*x_n*3 + x4*3 + i];
		interpolated[i] = z00*(x4-x)*(y4-y) + z10*(x-x2)*(y2-y) + z01*(x3-x)*(y-y3) + z11*(x-x1)*(y-y1);
	}
}

// start[0] --> x-coordinate
// start[1] --> y-coordinate
// end[0] --> x-coordinate
// end[1] --> y-coordinate
__device__
float rmsd(int *X, int *Y, int start[], int end[], int angle, int q_m, int q_n, int x_m, int x_n) {
	int s_x = start[0];
	int s_y = start[1];
	int e_x = end[0];
	int e_y = end[1];
	int m = e_y - s_y;
	int n = e_x - s_x;
	float rmsd = 0;
	for(int i = s_y; i < e_y; i++){
		for(int j = s_x; j < e_x; j++){
			float interpolated[3];
			int point[2] = {j, i};
			interpolate(X, point, start, angle, interpolated, x_m, x_n);
			if(interpolated[0] == -1){
				return FLT_MAX;
			}
			for(int k = 0; k < 3; k++){
				rmsd += (interpolated[k] - Y[(i - s_y)*q_n*3 + (j - s_x)*3 + k])*(interpolated[k] - Y[(i - s_y)*q_n*3 + (j - s_x)*3 + k]);
			}	
		}
	}
	rmsd /= m*n*3;
	return sqrt(rmsd);
}

void get_m_n(string file, int &m, int &n){
	ifstream in;
	in.open(file);
	in >> m;
	in >> n;
	in.close();
}

void readImage(string file, int *A) {
	int m, n, pixel;

	ifstream in;
	in.open(file);
	in >> m;
	in >> n;

	for(int i = 0; i < m; i++){
		for(int j = 0; j < n; j++){
			for(int k = 0; k < 3; k++){
				in >> pixel;
				A[(m - 1 - i)*n*3 + j*3 + k] = pixel;
			}
		}
	}
	in.close();
}


void filterBox(int *res, int angle, int q_m, int q_n){
	int *pivot, *br, *tl, *tr;
	pivot = new int[2];
	br = new int[2];
	tl = new int[2];
	tr = new int[2];

	pivot[0] = 0; pivot[1] = 0;
	br[0] = q_n - 1; br[1] = 0;
	tl[0] = 0; tl[1] = q_m - 1;
	tr[0] = q_n - 1; tr[1] = q_m - 1;

	float *tl_n, *br_n, *tr_n;
	tl_n = new float[2];
	br_n = new float[2];
	tr_n = new float[2];
	getRotatedCoordinateHost(tl_n, tl, pivot, angle);
	getRotatedCoordinateHost(br_n, br, pivot, angle);
	getRotatedCoordinateHost(tr_n, tr, pivot, angle);

	int x1, x2, y1, y2;

	if(angle == 45){
		x1 = ceil(tl_n[0]);
		x2 = floor(br_n[0]);
		y1 = pivot[1];
		y2 = floor(tr_n[1]);
	}
	else if(angle == -45){
		x1 = pivot[0];
		x2 = floor(tr_n[0]);
		y1 = ceil(br_n[1]);
		y2 = floor(tl_n[1]);
	}
	else{
		x1 = pivot[0];
		x2 = br[0];
		y1 = pivot[1];
		y2 = tl[1];
	}
	res[0] = x1;
	res[1] = x2;
	res[2] = y1;
	res[3] = y2;
	delete pivot;
	delete br;
	delete tl;
	delete tr;
	delete tl_n;
	delete br_n;
	delete tr_n;
}

// box[0] --> left-x, box[1] --> right-x, box[2] --> bottom-y, box[3] --> top-y
// pivot[0] --> x-coordinate, pivot[1] --> y-coordinate
__device__
float filter(float *grey_data_img, float query_grey, int box[], int pivot[], float t2, int x_m, int x_n){
	// return true;

	int x1 = pivot[0] + box[0];
	int x2 = pivot[0] + box[1];
	int y1 = pivot[1] + box[2];
	int y2 = pivot[1] + box[3];
	if(!inLimits(x1, y1, x_n, x_m) || !inLimits(x2, y2, x_n, x_m)){
		return FLT_MAX; // Actually false
	}

	float data_grey = 0;
	int count = 0;
	for(int i = y1; i <= y2; i++){
		for(int j = x1; j <= x2; j++){
			if(inLimits(j, i, x_n, x_m)){
				data_grey += (float)grey_data_img[i*x_n + j]; // Check this?
				count++;
			}
		}
	}
	data_grey /= (x2 - x1 + 1)*(y2 - y1 + 1);
	return abs(data_grey - query_grey);
}

void getGrey(int *X, float *grey_data_img, int x_m, int x_n){
	for(int i = 0; i < x_m; i++){
		for(int j = 0; j < x_n; j++){
			float x = 0;
			for(int k = 0; k < 3; k++){
				x += X[i*x_n*3 + j*3 + k];
			}
			grey_data_img[i*x_n + j] = (float)x/3;
		}
	}
}

__global__
void kernel(float *res, int *X, int *Q, float *grey_data_img, int *box0, int *box1, int *box2, int x_m, int x_n, int q_m, int q_n, float t1, float t2, float query_grey) {
	int angles[3] = {0, 45, -45};
	int offset = blockDim.x*blockIdx.x + threadIdx.x;
	if(offset >= x_m*x_n){
		return;
	}
	int i = offset / x_n; // y-coordinate
	int j = offset % x_n; // x-coordinate
	int pivot[2] = {j, i};
	for(int iter = 0; iter < 3; ++iter){
		int angle = angles[iter];
		res[i*x_n*3 + j*3 + iter] = -1;
		int box[4];
		if(iter == 0){
			box[0] = box0[0]; box[1] = box0[1]; box[2] = box0[2]; box[3] = box0[3];
		}
		else if(iter == 1){
			box[0] = box1[0]; box[1] = box1[1]; box[2] = box1[2]; box[3] = box1[3];
		}
		else{
			box[0] = box2[0]; box[1] = box2[1]; box[2] = box2[2]; box[3] = box2[3];
		}
		float diff = filter(grey_data_img, query_grey, box, pivot, t2, x_m, x_n);
		if(diff <= t2){
			int start[2] = {j, i};
			int end[2] = {j + q_n, i + q_m};
			float f = rmsd(X, Q, start, end, angle, q_m, q_n, x_m, x_n);
			if(f <= t1){
				res[i*x_n*3 + j*3 + iter] = (float)f;
			}
		}
	}
}

int main(int argc, char *argv[]) {
	// auto begin = std::chrono::high_resolution_clock::now();
	string data_image_path = argv[1];
	string query_image_path = argv[2];
	float t1 = stof(argv[3]);
	float t2 = stof(argv[4]);
	int topn = stoi(argv[5]);

	int x_m, x_n, q_m, q_n; // q_m is y-max, q_n is x-max, x_m is y-max, x_n is x-max

	int *X, *Q, *dX, *dQ;
	float *grey_data_img, *res, *dgrey_data_img, *dres;

	get_m_n(data_image_path, x_m, x_n);
	X = new int[x_m*x_n*3];
	res = new float[x_m*x_n*3];
	readImage(data_image_path, X);

	get_m_n(query_image_path, q_m, q_n);
	Q = new int[q_m*q_n*3];
	readImage(query_image_path, Q);

	grey_data_img = new float[x_m*x_n];
	getGrey(X, grey_data_img, x_m, x_n);

	// Finding Average Greyscale value of Query Image
	float query_grey = 0;
	for(int i = 0; i < q_m; i++){
		for(int j = 0; j < q_n; j++){
			float x = 0;
			for(int k = 0; k < 3; k++){
				x += Q[i*q_n*3 + j*3 + k];
			}
			query_grey += (float)x/3;
		}
	}
	query_grey /= q_m*q_n;

	// Finding axis-aligned bounding box
	int *boxes[3];
	int *dbox0, *dbox1, *dbox2;
	for(int i = 0; i < 3; i++){
		boxes[i] = new int[4];
	}
	filterBox(boxes[0], 0, q_m, q_n);
	filterBox(boxes[1], 45, q_m, q_n);
	filterBox(boxes[2], -45, q_m, q_n);

	// Allocate on GPU
	hipMalloc(&dX, x_m*x_n*3*sizeof(int));
	hipMalloc(&dQ, q_m*q_n*3*sizeof(int));
	hipMalloc(&dgrey_data_img, x_m*x_n*sizeof(float));
	hipMalloc(&dres, x_m*x_n*3*sizeof(float));
	hipMalloc(&dbox0, 4*sizeof(int));
	hipMalloc(&dbox1, 4*sizeof(int));
	hipMalloc(&dbox2, 4*sizeof(int));

	// Copy on GPU
	hipMemcpy(dX, X, x_m*x_n*3*sizeof(int), hipMemcpyDefault);
	hipMemcpy(dQ, Q, q_m*q_n*3*sizeof(int), hipMemcpyDefault);
	hipMemcpy(dgrey_data_img, grey_data_img, x_m*x_n*sizeof(float), hipMemcpyDefault);
	hipMemcpy(dbox0, boxes[0], 4*sizeof(int), hipMemcpyDefault);
	hipMemcpy(dbox1, boxes[1], 4*sizeof(int), hipMemcpyDefault);
	hipMemcpy(dbox2, boxes[2], 4*sizeof(int), hipMemcpyDefault);

	int num_blocks = ceil((float)(x_n*x_m)/1024);
	kernel<<<num_blocks, 1024>>>(dres, dX, dQ, dgrey_data_img, dbox0, dbox1, dbox2, x_m, x_n, q_m, q_n, t1, t2, query_grey);

	hipMemcpy(res, dres, x_m*x_n*3*sizeof(float), hipMemcpyDeviceToHost);
	int angles[3] = {0, 45, -45};

	priority_queue<pair<float, tuple<int, int, int>>> q; 

	for(int i = 0; i < x_m; i++){ // i correcposds to y-coordinate
		for(int j = 0; j < x_n; j++){ // j corresponds to x-coordinate
			for(int k = 0; k < 3; k++){
				float f = res[i*x_n*3 + j*3 + k];
				if(f != -1){
					if(q.size() >= topn && f > q.top().first){
						continue;
					}
					q.push({f, {i, j, angles[k]}});
					if(q.size() > topn){
						q.pop();
					}
				}
			}
		}
	}
	
	ofstream outdata;
	outdata.open("output.txt");
	priority_queue<pair<float, tuple<int, int, int>>> output; 
	while(!q.empty()){
		pair<float, tuple<int, int, int>> p = q.top();
		tuple<int, int, int> tup = p.second;
		q.pop();
		output.push({-p.first, tup});
	}
	while(!output.empty()){
		pair<float, tuple<int, int, int>> p = output.top();
		output.pop();
		tuple<int, int, int> tup = p.second;
		//cout << get<0>(tup) << " " << get<1>(tup) << " " << get<2>(tup) << " " << -1*p.first << endl; // Remove this
		outdata << get<0>(tup) << " " << get<1>(tup) << " " << get<2>(tup) << endl;
	}

	delete X;
	delete Q;
	delete grey_data_img;

	hipFree(dX);
	hipFree(dQ);
	hipFree(dgrey_data_img);
	hipFree(dres);
	hipFree(dbox0);
	hipFree(dbox1);
	hipFree(dbox2);

	outdata.close();
	//auto end1 = std::chrono::high_resolution_clock::now();
	//cout << "Time taken for completion: " << (1e-6 * (std::chrono::duration_cast<std::chrono::nanoseconds>(end1 - begin)).count()) << "ms" << endl;
	// Remove this
}